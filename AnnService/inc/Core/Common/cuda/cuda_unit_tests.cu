
#include <hip/hip_runtime.h>
/*
#include "Refine.hxx"

#define T_BLOCKS 1
#define T_THREADS 64

__device__ void printList(ListElt<int>* list, int size) {
  if(threadIdx.x==0 && blockIdx.x==0) {
    printf("size:%d\n", size);
  for(int i=0; i<size; i++) {
    printf("%d, %d, %d\n", list[i].id, list[i].dist, list[i].checkedFlag);
  }
  printf("\n");
  }
  __syncthreads();
}

__global__ void test_removeDuplicates(ListElt<int>* listMem) {
  ListElt<int>* list = &listMem[blockIdx.x*LISTSIZE*2];

  typedef cub::BlockRadixSort<int, T_THREADS, LISTCAP/T_THREADS, ListElt<int>> BlockRadixSortT;
__shared__ typename BlockRadixSortT::TempStorage temp_storage;
__shared__ int border_vals[T_THREADS];


  for(int i=threadIdx.x; i<LISTSIZE; i+=blockDim.x) {
    list[i].id = i;
    list[i+LISTSIZE].id = i;
    list[i].dist = -i;
    list[i+LISTSIZE].dist = -i;
    list[i].checkedFlag=true;
    list[i+LISTSIZE].checkedFlag=false;
  }
  __syncthreads();


  int listSize = LISTSIZE*2;

  __syncthreads();
  printList(list, listSize);
  __syncthreads();

  sortListById<int, int, 0, T_THREADS>(list, &listSize, &temp_storage);

  removeDuplicatesAndCompact<int, int, 0, T_THREADS>(list, &listSize, &temp_storage, border_vals);


  printList(list, listSize);
}


int main(int argc, char* argv[]) {

  ListElt<int>* listMem;
  cudaMalloc(&listMem, T_BLOCKS*LISTSIZE*2*sizeof(ListElt<int>));

  test_removeDuplicates<<<T_BLOCKS, T_THREADS>>>(listMem);
  printf("Error: %d\n", cudaDeviceSynchronize());
}

*/
